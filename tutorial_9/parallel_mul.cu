#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 8000000
#define THREADS_PER_BLOCK 1024

__global__ void vector_mul(double *a, double *b, double *c){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N){
        c[index] = a[index] * b[index];
    }
}

int main(){
    FILE *input1 = fopen("file1.txt", "r");
    FILE *input2 = fopen("file2.txt", "r");

    double *a = (double*)malloc(N * sizeof(double));
    double *b = (double*)malloc(N * sizeof(double));
    double *c_mul = (double*)malloc(N * sizeof(double));

    for (int i = 0; i < N; i++){
        fscanf(input1, "%lf", &a[i]);
        fscanf(input2, "%lf", &b[i]);
    }

    fclose(input1);
    fclose(input2);

    double *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, N * sizeof(double));
    hipMalloc((void **)&d_b, N * sizeof(double));
    hipMalloc((void **)&d_c, N * sizeof(double));

    hipMemcpy(d_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipEventRecord(start);
    vector_mul<<<blocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(c_mul, d_c, N * sizeof(double), hipMemcpyDeviceToHost);

    printf("Time taken by parallel multiplication code: %f ms\n", milliseconds);

    // Print the first 10 values of the result
    printf("First 10 values of the result:\n");
    for (int i = 0; i < 10; i++){
        printf("%f\n", c_mul[i]);
    }
    
    free(a);
    free(b);
    free(c_mul);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}